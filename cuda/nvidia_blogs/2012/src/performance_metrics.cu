#include "hip/hip_runtime.h"
/*
   Link:
   https://developer.nvidia.com/blog/how-implement-performance-metrics-cuda-cc/
*/

#include <cstdio>

#include <hip/hip_runtime_api.h>

__global__ void saxpy(int n, float a, float *x, float *y) {
  unsigned int i{blockIdx.x * blockDim.x + threadIdx.x};
  if (i < n)
    y[i] = a * x[i] + y[i];
}

int main(int argc, char *argv[]) {
  constexpr int N{80 * (1 << 20)};
  float *d_x, *d_y;
  float *x{new float[N]};
  float *y{new float[N]};

  const size_t bytes{N * sizeof(float)};
  hipMalloc(&d_x, bytes);
  hipMalloc(&d_y, bytes);

  for (int i = 0; i < N; ++i) {
    x[i] = 1.0f;
    y[i] = 2.0f;
  }

  hipMemcpy(d_x, x, bytes, hipMemcpyHostToDevice);
  hipMemcpy(d_y, y, bytes, hipMemcpyHostToDevice);

  hipEvent_t start, stop;
  hipEventCreate(&start);
  hipEventCreate(&stop);
  hipEventRecord(start);

  constexpr int block_size{512};
  saxpy<<<(N + block_size - 1) / block_size, block_size>>>(N, 2.0f, d_x, d_y);

  hipEventRecord(stop);
  hipEventSynchronize(stop);

  hipMemcpy(y, d_y, bytes, hipMemcpyDeviceToHost);

  float milliseconds{0};
  hipEventElapsedTime(&milliseconds, start, stop);

  float max_error{0.0f};
  for (int i = 0; i < N; ++i)
    max_error = max(max_error, std::abs(y[i] - 4.0f));
  std::printf("Max error: %f\n", max_error);

  /*
    2 reads (x and y) and 1 write (y)
    RTX 4090: Effective Bandwidth (GB/s): 885.247777
  */
  std::printf("Effective Bandwidth (GB/s): %f\n",
              3 * bytes / (milliseconds * 1e6));

  hipEventDestroy(start);
  hipEventDestroy(stop);
  hipFree(d_x);
  hipFree(d_y);
  delete[] x;
  delete[] y;

  return 0;
}
