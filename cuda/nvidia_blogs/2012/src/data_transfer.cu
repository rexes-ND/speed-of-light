/*
    Link: https://developer.nvidia.com/blog/how-optimize-data-transfers-cuda-cc/
*/


#include <hip/hip_runtime.h>
#include <cassert>
#include <cstdio>
#include <cstring>

#include <hip/hip_runtime_api.h>

inline hipError_t checkCuda(hipError_t result) {
  if (result != hipSuccess) {
    std::fprintf(stderr, "CUDA Runtime Error: %s\n",
                 hipGetErrorString(result));
    assert(result == hipSuccess);
  }
  return result;
}

void profileCopies(float *h_a, float *h_b, float *d, unsigned int n,
                   const char *desc) {
  /*
    Profiles `h_a` -> `d` and `d` -> `h_b`.
  */
  std::printf("\n%s transfer\n", desc);
  const size_t bytes{n * sizeof(float)};

  hipEvent_t startEvent, stopEvent;
  checkCuda(hipEventCreate(&startEvent));
  checkCuda(hipEventCreate(&stopEvent));

  checkCuda(hipEventRecord(startEvent, 0));
  checkCuda(hipMemcpy(d, h_a, bytes, hipMemcpyHostToDevice));
  checkCuda(hipEventRecord(stopEvent, 0));
  checkCuda(hipEventSynchronize(stopEvent));

  float time{};
  checkCuda(hipEventElapsedTime(&time, startEvent, stopEvent));
  std::printf("\tHost to Device bandwidth (GB/s): %f\n", bytes * 1e-6 / time);

  checkCuda(hipEventRecord(startEvent, 0));
  checkCuda(hipMemcpy(h_b, d, bytes, hipMemcpyDeviceToHost));
  checkCuda(hipEventRecord(stopEvent, 0));
  checkCuda(hipEventSynchronize(stopEvent));

  checkCuda(hipEventElapsedTime(&time, startEvent, stopEvent));
  std::printf("\tDevice to Host bandwidth (GB/s): %f\n", bytes * 1e-6 / time);

  for (int i = 0; i < n; ++i) {
    if (h_a[i] != h_b[i]) {
      std::printf("*** %s transfers failed ***\n", desc);
      break;
    }
  }
}

int main() {
  /*
      CMD: nsys profile --stats=true -o /dev/null ./build/data_transfer
  */
  //   constexpr unsigned int N{1 << 20};
  //   int *h_a{new int[N]};
  //   int *d_a;
  //   const size_t bytes{N * sizeof(int)};
  //   cudaMalloc(&d_a, bytes);

  //   std::memset(h_a, 0, bytes);
  //   cudaMemcpy(d_a, h_a, bytes, cudaMemcpyHostToDevice);
  //   cudaMemcpy(h_a, d_a, bytes, cudaMemcpyDeviceToHost);

  /*
    Device: NVIDIA GeForce RTX 4090
    Transfer size (MB): 16

    Pageable transfer
        Host to Device bandwidth (GB/s): 6.159978
        Device to Host bandwidth (GB/s): 5.782758

    Pinned transfer
        Host to Device bandwidth (GB/s): 15.100895
        Device to Host bandwidth (GB/s): 12.688480
  */

  constexpr unsigned int nElements{4 << 20};
  const size_t bytes{nElements * sizeof(float)};

  float *h_aPageable{new float[nElements]};
  float *h_bPageable{new float[nElements]};

  float *h_aPinned, *h_bPinned;
  float *d_a;
  checkCuda(hipHostMalloc(&h_aPinned, bytes, hipHostMallocDefault));
  checkCuda(hipHostMalloc(&h_bPinned, bytes, hipHostMallocDefault));
  checkCuda(hipMalloc(&d_a, bytes));

  for (int i = 0; i < nElements; ++i)
    h_aPageable[i] = i;

  std::memcpy(h_aPinned, h_aPageable, bytes);
  std::memset(h_bPageable, 0, bytes);
  std::memset(h_bPinned, 0, bytes);

  hipDeviceProp_t prop;
  checkCuda(hipGetDeviceProperties(&prop, 0));

  std::printf("\nDevice: %s\n", prop.name);
  std::printf("Transfer size (MB): %lu\n", bytes >> 20);

  profileCopies(h_aPageable, h_bPageable, d_a, nElements, "Pageable");
  profileCopies(h_aPinned, h_bPinned, d_a, nElements, "Pinned");
  std::printf("\n");

  hipFree(d_a);
  hipHostFree(h_aPinned);
  hipHostFree(h_bPinned);
  delete[] h_aPageable;
  delete[] h_bPageable;

  return 0;
}
