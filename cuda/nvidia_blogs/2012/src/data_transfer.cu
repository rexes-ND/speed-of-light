
#include <hip/hip_runtime.h>
#include <algorithm>
#include <iostream>
#include <string>
#include <vector>

#include <hip/hip_runtime_api.h>

#define CHECK_CUDA_ERROR(err) __check_cuda_error(err, __FILE__, __LINE__)
static void __check_cuda_error(hipError_t err, const char *filename,
                               int line) {
  if (hipSuccess != err) {
    std::cerr << "CUDA API error: " << hipGetErrorString(err) << " from file "
              << filename << ", line " << line << std::endl;
    exit(err);
  }
}

void profile_copy(float *h_a, float *h_b, float *d, unsigned int n,
                  const std::string &desc) {
  // 1. copy `h_a` to `d`
  // 2. copy `d`   to `h_b`
  std::cout << std::endl << desc << " transfer" << std::endl;
  const auto bytes = n * sizeof(float);

  hipEvent_t start_event, stop_event;
  CHECK_CUDA_ERROR(hipEventCreate(&start_event));
  CHECK_CUDA_ERROR(hipEventCreate(&stop_event));

  CHECK_CUDA_ERROR(hipEventRecord(start_event));
  CHECK_CUDA_ERROR(hipMemcpy(d, h_a, bytes, hipMemcpyDefault));
  CHECK_CUDA_ERROR(hipEventRecord(stop_event));
  CHECK_CUDA_ERROR(hipEventSynchronize(stop_event));

  float time;
  CHECK_CUDA_ERROR(hipEventElapsedTime(&time, start_event, stop_event));
  std::cout << "\tHost to Device bandwidth (GB/s): " << bytes * 1e-6 / time
            << std::endl;

  CHECK_CUDA_ERROR(hipEventRecord(start_event));
  CHECK_CUDA_ERROR(hipMemcpy(h_b, d, bytes, hipMemcpyDefault));
  CHECK_CUDA_ERROR(hipEventRecord(stop_event));
  CHECK_CUDA_ERROR(hipEventSynchronize(stop_event));

  CHECK_CUDA_ERROR(hipEventElapsedTime(&time, start_event, stop_event));
  std::cout << "\tDevice to Host bandwidth (GB/s): " << bytes * 1e-6 / time
            << std::endl;

  for (int i = 0; i < n; ++i) {
    if (h_a[i] != h_b[i]) {
      std::cout << "*** " << desc << " transfers failed ***" << std::endl;
      break;
    }
  }
}

int main() {
  /*
    // nsys profile --stats=true -o /dev/null <executable>

    constexpr auto N = 1U << 20;
    std::vector<int> h_a(N);

    int *d_a;
    const auto bytes = N * sizeof(int);
    cudaMalloc(&d_a, bytes);

    cudaMemcpy(d_a, h_a.data(), bytes, cudaMemcpyDefault);
    cudaMemcpy(h_a.data(), d_a, bytes, cudaMemcpyDefault);
  */

  constexpr auto N = 4U << 20;
  constexpr auto bytes = N * sizeof(float);

  std::vector<float> h_a(N);
  std::vector<float> h_b(N);

  float *h_a_pinned, *h_b_pinned;
  CHECK_CUDA_ERROR(hipHostMalloc(&h_a_pinned, bytes, hipHostMallocDefault));
  CHECK_CUDA_ERROR(hipHostMalloc(&h_b_pinned, bytes, hipHostMallocDefault));

  float *d_a;
  CHECK_CUDA_ERROR(hipMalloc(&d_a, bytes));

  for (int i = 0; i < N; ++i)
    h_a[i] = i;

  std::copy_n(h_a.data(), N, h_a_pinned);
  std::fill(h_b_pinned, h_b_pinned + N, 0);

  hipDeviceProp_t prop;
  CHECK_CUDA_ERROR(hipGetDeviceProperties(&prop, 0));

  std::cout << std::endl << "Device: " << prop.name << std::endl;
  std::cout << "Transfer size (MB): " << (bytes >> 20) << std::endl;

  profile_copy(h_a.data(), h_b.data(), d_a, N, "Pageable");
  profile_copy(h_a_pinned, h_b_pinned, d_a, N, "Pinned");
  std::cout << std::endl;

  hipFree(d_a);
  hipHostFree(h_a_pinned);
  hipHostFree(h_b_pinned);

  return 0;
}
