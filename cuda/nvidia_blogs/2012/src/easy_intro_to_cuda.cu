#include "hip/hip_runtime.h"
/*
    Link: https://developer.nvidia.com/blog/easy-introduction-cuda-c-and-c/
*/

#include <cstdio>

#include <hip/hip_runtime_api.h>

__global__ void saxpy(int n, float a, float *x, float *y) {
  unsigned int i{blockIdx.x * blockDim.x + threadIdx.x};
  if (i < n)
    y[i] = a * x[i] + y[i];
}

int main() {
  constexpr int N{1 << 20};
  float *d_x, *d_y;
  float *x{new float[N]};
  float *y{new float[N]};

  const size_t bytes{N * sizeof(float)};
  hipMalloc(&d_x, bytes);
  hipMalloc(&d_y, bytes);

  for (int i = 0; i < N; ++i) {
    x[i] = 1.0f;
    y[i] = 2.0f;
  }

  hipMemcpy(d_x, x, bytes, hipMemcpyHostToDevice);
  hipMemcpy(d_y, y, bytes, hipMemcpyHostToDevice);

  constexpr int block_size{256};
  saxpy<<<(N + block_size - 1) / block_size, block_size>>>(N, 2.0f, d_x, d_y);

  hipMemcpy(y, d_y, bytes, hipMemcpyDeviceToHost);

  float max_error{0.0f};
  for (int i = 0; i < N; ++i)
    max_error = max(max_error, std::abs(y[i] - 4.0f));
  std::printf("Max error: %f\n", max_error);

  hipFree(d_x);
  hipFree(d_y);
  delete[] x;
  delete[] y;

  return 0;
}
