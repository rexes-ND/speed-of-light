/*
  Link: https://developer.nvidia.com/blog/how-overlap-data-transfers-cuda-cc/
*/


#include <hip/hip_runtime.h>
#include <array>
#include <cassert>
#include <cstdio>
#include <cstring>

#include <hip/hip_runtime_api.h>

inline hipError_t checkCuda(hipError_t result) {
  if (result != hipSuccess) {
    std::fprintf(stderr, "CUDA Runtime Error: %s\n",
                 hipGetErrorString(result));
    assert(result == hipSuccess);
  }
  return result;
}

__global__ void kernel(float *a, int offset) {
  auto i{offset + threadIdx.x + blockIdx.x * blockDim.x};
  auto x{static_cast<float>(i)};
  auto s{sinf(x)};
  auto c{cosf(x)};
  a[i] += sqrtf(s * s + c * c);
}

float maxError(float *a, int n) {
  float maxE{0};
  for (int i = 0; i < n; ++i) {
    float error{fabs(a[i] - 1.0f)};
    if (error > maxE)
      maxE = error;
  }
  return maxE;
}

int main(int argc, char **argv) {
  constexpr int blockSize{256};
  constexpr int nStreams{4};
  constexpr int n{4 * 1024 * blockSize * nStreams};
  constexpr int streamSize{n / nStreams};
  constexpr int bytes{n * sizeof(float)};
  constexpr int streamBytes{streamSize * sizeof(float)};

  int devId{(argc > 1) ? atoi(argv[1]) : 0};

  hipDeviceProp_t prop;
  checkCuda(hipGetDeviceProperties(&prop, devId));
  printf("Device : %s\n", prop.name);
  checkCuda(hipSetDevice(devId));

  float *a, *d_a;
  checkCuda(hipHostMalloc(&a, bytes, hipHostMallocDefault));
  checkCuda(hipMalloc(&d_a, bytes));

  hipEvent_t startEvent, stopEvent, dummyEvent;
  std::array<hipStream_t, nStreams> streams;
  checkCuda(hipEventCreate(&startEvent));
  checkCuda(hipEventCreate(&stopEvent));
  checkCuda(hipEventCreate(&dummyEvent));
  for (int i = 0; i < nStreams; ++i)
    checkCuda(hipStreamCreate(&streams[i]));

  // baseline case - sequential transfer and execute
  std::memset(a, 0, bytes);
  checkCuda(hipEventRecord(startEvent, 0));
  checkCuda(hipMemcpy(d_a, a, bytes, hipMemcpyHostToDevice));
  kernel<<<n / blockSize, blockSize>>>(d_a, 0);
  checkCuda(hipMemcpy(a, d_a, bytes, hipMemcpyDeviceToHost));
  checkCuda(hipEventRecord(stopEvent, 0));
  checkCuda(hipEventSynchronize(stopEvent));
  float ms;
  checkCuda(hipEventElapsedTime(&ms, startEvent, stopEvent));
  std::printf("Time for sequential transfer and execute (ms): %f\n", ms);
  std::printf("\tmax error: %e\n", maxError(a, n));

  // async v1: loop over {copy, kernel, copy}
  std::memset(a, 0, bytes);
  checkCuda(hipEventRecord(startEvent, 0));
  for (int i = 0; i < nStreams; ++i) {
    const int offset{i * streamSize};
    checkCuda(hipMemcpyAsync(&d_a[offset], &a[offset], streamBytes,
                              hipMemcpyHostToDevice, streams[i]));
    kernel<<<streamSize / blockSize, blockSize, 0, streams[i]>>>(d_a, offset);
    checkCuda(hipMemcpyAsync(&a[offset], &d_a[offset], streamBytes,
                              hipMemcpyDeviceToHost, streams[i]));
  }
  checkCuda(hipEventRecord(stopEvent, 0));
  checkCuda(hipEventSynchronize(stopEvent));
  checkCuda(hipEventElapsedTime(&ms, startEvent, stopEvent));
  std::printf("Time for asynchronous V1 transfer and execute (ms): %f\n", ms);
  std::printf("\tmax error: %e\n", maxError(a, n));

  // async v2: loop over copy, loop over kernel, loop over copy
  std::memset(a, 0, bytes);
  checkCuda(hipEventRecord(startEvent, 0));
  for (int i = 0; i < nStreams; ++i) {
    const int offset{i * streamSize};
    checkCuda(hipMemcpyAsync(&d_a[offset], &a[offset], streamBytes,
                              hipMemcpyHostToDevice, streams[i]));
  }
  for (int i = 0; i < nStreams; ++i) {
    const int offset{i * streamSize};
    kernel<<<streamSize / blockSize, blockSize, 0, streams[i]>>>(d_a, offset);
  }
  for (int i = 0; i < nStreams; ++i) {
    const int offset{i * streamSize};
    checkCuda(hipMemcpyAsync(&a[offset], &d_a[offset], streamBytes,
                              hipMemcpyDeviceToHost, streams[i]));
  }
  checkCuda(hipEventRecord(stopEvent, 0));
  checkCuda(hipEventSynchronize(stopEvent));
  checkCuda(hipEventElapsedTime(&ms, startEvent, stopEvent));
  std::printf("Time for asynchronouse V2 transfer and execute (ms): %f\n", ms);
  std::printf("\tmax error: %e\n", maxError(a, n));

  // Cleanup
  checkCuda(hipEventDestroy(startEvent));
  checkCuda(hipEventDestroy(stopEvent));
  checkCuda(hipEventDestroy(dummyEvent));
  for (int i = 0; i < nStreams; ++i)
    checkCuda(hipStreamDestroy(streams[i]));
  hipFree(d_a);
  hipHostFree(a);

  return 0;
}
