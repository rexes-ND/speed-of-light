
#include <hip/hip_runtime.h>
#include <algorithm>
#include <array>
#include <iostream>

#include <hip/hip_runtime_api.h>

#define CHECK_CUDA_ERROR(err) __check_cuda_error(err, __FILE__, __LINE__)
static void __check_cuda_error(hipError_t err, const char *filename,
                               int line) {
  if (hipSuccess != err) {
    std::cerr << "CUDA API error: " << hipGetErrorString(err) << " from file "
              << filename << ", line " << line << std::endl;
    exit(err);
  }
}

__global__ void kernel(float *a, int offset) {
  const auto i = offset + threadIdx.x + blockIdx.x * blockDim.x;
  const auto x = static_cast<float>(i);
  const auto s = sinf(x);
  const auto c = cosf(x);
  a[i] += sqrtf(s * s + c * c);
}

float maxError(float *a, int n) {
  float max_error = 0;
  for (int i = 0; i < n; ++i)
    max_error = std::max(max_error, std::abs(a[i] - 1.0f));
  return max_error;
}

int main(int argc, char **argv) {
  constexpr auto block_size = 256U;
  constexpr auto num_streams = 4;
  constexpr auto N = 4 * 1024 * block_size * num_streams;
  constexpr auto stream_size = N / num_streams;
  constexpr auto bytes = N * sizeof(float);
  constexpr auto stream_bytes = stream_size * sizeof(float);

  const auto dev_id = (argc > 1) ? atoi(argv[1]) : 0;

  hipDeviceProp_t prop;
  CHECK_CUDA_ERROR(hipGetDeviceProperties(&prop, dev_id));
  std::cout << "Device : " << prop.name << std::endl;
  CHECK_CUDA_ERROR(hipSetDevice(dev_id));

  float *a, *d_a;
  CHECK_CUDA_ERROR(hipHostMalloc(&a, bytes, hipHostMallocDefault));
  CHECK_CUDA_ERROR(hipMalloc(&d_a, bytes));

  hipEvent_t startEvent, stopEvent, dummyEvent;
  std::array<hipStream_t, num_streams> streams;
  CHECK_CUDA_ERROR(hipEventCreate(&startEvent));
  CHECK_CUDA_ERROR(hipEventCreate(&stopEvent));
  CHECK_CUDA_ERROR(hipEventCreate(&dummyEvent));
  for (int i = 0; i < num_streams; ++i)
    CHECK_CUDA_ERROR(hipStreamCreate(&streams[i]));

  // baseline case - sequential transfer and execute
  std::fill_n(a, N, 0);
  CHECK_CUDA_ERROR(hipEventRecord(startEvent));
  CHECK_CUDA_ERROR(hipMemcpy(d_a, a, bytes, hipMemcpyDefault));
  kernel<<<N / block_size, block_size>>>(d_a, 0);
  CHECK_CUDA_ERROR(hipMemcpy(a, d_a, bytes, hipMemcpyDefault));
  CHECK_CUDA_ERROR(hipEventRecord(stopEvent));
  CHECK_CUDA_ERROR(hipEventSynchronize(stopEvent));
  float ms;
  CHECK_CUDA_ERROR(hipEventElapsedTime(&ms, startEvent, stopEvent));
  std::cout << "Time for sequential transfer and execute (ms): " << ms
            << std::endl;
  std::cout << "\tmax error: " << maxError(a, N) << std::endl;

  // async v1: loop over {copy, kernel, copy}
  std::fill_n(a, N, 0);
  CHECK_CUDA_ERROR(hipEventRecord(startEvent, 0));
  for (int i = 0; i < num_streams; ++i) {
    const auto offset = i * stream_size;
    CHECK_CUDA_ERROR(hipMemcpyAsync(&d_a[offset], &a[offset], stream_bytes,
                                     hipMemcpyDefault, streams[i]));
    kernel<<<stream_size / block_size, block_size, 0, streams[i]>>>(d_a,
                                                                    offset);
    CHECK_CUDA_ERROR(hipMemcpyAsync(&a[offset], &d_a[offset], stream_bytes,
                                     hipMemcpyDefault, streams[i]));
  }
  CHECK_CUDA_ERROR(hipEventRecord(stopEvent, 0));
  CHECK_CUDA_ERROR(hipEventSynchronize(stopEvent));
  CHECK_CUDA_ERROR(hipEventElapsedTime(&ms, startEvent, stopEvent));
  std::cout << "Time for asynchronous V1 transfer and execute (ms): " << ms
            << std::endl;
  std::cout << "\tmax error: " << maxError(a, N) << std::endl;

  // async v2: loop over copy, loop over kernel, loop over copy
  // std::memset(a, 0, bytes);
  std::fill_n(a, N, 0);
  CHECK_CUDA_ERROR(hipEventRecord(startEvent, 0));
  for (int i = 0; i < num_streams; ++i) {
    const auto offset = i * stream_size;
    CHECK_CUDA_ERROR(hipMemcpyAsync(&d_a[offset], &a[offset], stream_bytes,
                                     hipMemcpyDefault, streams[i]));
  }
  for (int i = 0; i < num_streams; ++i) {
    const auto offset = i * stream_size;
    kernel<<<stream_size / block_size, block_size, 0, streams[i]>>>(d_a,
                                                                    offset);
  }
  for (int i = 0; i < num_streams; ++i) {
    const auto offset = i * stream_size;
    CHECK_CUDA_ERROR(hipMemcpyAsync(&a[offset], &d_a[offset], stream_bytes,
                                     hipMemcpyDeviceToHost, streams[i]));
  }
  CHECK_CUDA_ERROR(hipEventRecord(stopEvent, 0));
  CHECK_CUDA_ERROR(hipEventSynchronize(stopEvent));
  CHECK_CUDA_ERROR(hipEventElapsedTime(&ms, startEvent, stopEvent));
  std::cout << "Time for asynchronouse V2 transfer and execute (ms): " << ms
            << std::endl;
  std::cout << "\tmax error: " << maxError(a, N) << std::endl;

  // Cleanup
  CHECK_CUDA_ERROR(hipEventDestroy(startEvent));
  CHECK_CUDA_ERROR(hipEventDestroy(stopEvent));
  CHECK_CUDA_ERROR(hipEventDestroy(dummyEvent));
  for (int i = 0; i < num_streams; ++i)
    CHECK_CUDA_ERROR(hipStreamDestroy(streams[i]));
  hipFree(d_a);
  hipHostFree(a);

  return 0;
}
