
#include <hip/hip_runtime.h>
#include <iostream>
#include <string>
#include <vector>

#include <hip/hip_runtime_api.h>

#define CHECK_CUDA_ERROR(err) __check_cuda_error(err, __FILE__, __LINE__)
static void __check_cuda_error(hipError_t err, const char *filename,
                               int line) {
  if (hipSuccess != err) {
    std::cerr << "CUDA API error: " << hipGetErrorString(err) << " from file "
              << filename << ", line " << line << std::endl;
    exit(err);
  }
}

constexpr auto TILE_DIM = 32U;
constexpr auto BLOCK_ROWS = 8U;
constexpr auto NUM_REPS = 100U;

// Check errors and print GB/s
void postprocess(const float *ref, const float *res, uint n, float ms) {
  auto passed = true;
  for (int i = 0; i < n; ++i) {
    if (res[i] != ref[i]) {
      std::printf("%d %f %f\n", i, res[i], ref[i]);
      std::printf("%25s\n", "*** FAILED ***");
      passed = false;
      break;
    }
  }
  if (passed)
    std::printf("%20.2f\n", 2 * n * sizeof(float) * 1e-6 * NUM_REPS / ms);
}

// simple copy kernel
// Used as reference case representing best effective bandwidth
__global__ void copy(float *out, const float *in) {
  const auto x = blockIdx.x * TILE_DIM + threadIdx.x;
  const auto y = blockIdx.y * TILE_DIM + threadIdx.y;
  const auto width = gridDim.x * TILE_DIM;

  for (int j = 0; j < TILE_DIM; j += BLOCK_ROWS)
    out[(y + j) * width + x] = in[(y + j) * width + x];
}

// copy kernel using shared memory
// Also used as reference case, demonstrating effect of using shared memory.
__global__ void copySharedMem(float *out, const float *in) {
  __shared__ float tile[TILE_DIM * TILE_DIM];

  const auto x = blockIdx.x * TILE_DIM + threadIdx.x;
  const auto y = blockIdx.y * TILE_DIM + threadIdx.y;
  const auto width = gridDim.x * TILE_DIM;

  for (int j = 0; j < TILE_DIM; j += BLOCK_ROWS)
    tile[(threadIdx.y + j) * TILE_DIM + threadIdx.x] = in[(y + j) * width + x];

  // Included to mimic the transpose behavior.
  __syncthreads();

  for (int j = 0; j < TILE_DIM; j += BLOCK_ROWS)
    out[(y + j) * width + x] = tile[(threadIdx.y + j) * TILE_DIM + threadIdx.x];
}

// naive transpose
// Simplest transpose; doesn't use shared memory.
// Global memory reads are coalesced but writes are not.
__global__ void transposeNaive(float *out, const float *in) {
  const auto x = blockIdx.x * TILE_DIM + threadIdx.x;
  const auto y = blockIdx.y * TILE_DIM + threadIdx.y;
  const auto width = gridDim.x * TILE_DIM;

  for (int j = 0; j < TILE_DIM; j += BLOCK_ROWS)
    out[x * width + (y + j)] = in[(y + j) * width + x];
}

// coalesced transpose
// Uses shared memory to achieve coalescing in both reads and writes
// Tile width == #banks causes shared memory bank conflicts.
__global__ void transposeCoalesced(float *out, const float *in) {
  __shared__ float tile[TILE_DIM][TILE_DIM];

  auto x = blockIdx.x * TILE_DIM + threadIdx.x;
  auto y = blockIdx.y * TILE_DIM + threadIdx.y;
  const auto width = gridDim.x * TILE_DIM;

  for (int j = 0; j < TILE_DIM; j += BLOCK_ROWS)
    tile[threadIdx.y + j][threadIdx.x] = in[(y + j) * width + x];

  __syncthreads();

  x = blockIdx.y * TILE_DIM + threadIdx.x;
  y = blockIdx.x * TILE_DIM + threadIdx.y;

  for (int j = 0; j < TILE_DIM; j += BLOCK_ROWS)
    out[(y + j) * width + x] = tile[threadIdx.x][threadIdx.y + j];
}

// No bank-conflict transpose
// Same as transposeCoalesced except the first tile dimension is padded
// to avoid shared memory bank conflicts.
__global__ void transposeNoBankConflicts(float *out, const float *in) {
  __shared__ float tile[TILE_DIM][TILE_DIM + 1];

  auto x = blockIdx.x * TILE_DIM + threadIdx.x;
  auto y = blockIdx.y * TILE_DIM + threadIdx.y;
  const auto width = gridDim.x * TILE_DIM;

  for (int j = 0; j < TILE_DIM; j += BLOCK_ROWS)
    tile[threadIdx.y + j][threadIdx.x] = in[(y + j) * width + x];

  __syncthreads();

  x = blockIdx.y * TILE_DIM + threadIdx.x;
  y = blockIdx.x * TILE_DIM + threadIdx.y;

  for (int j = 0; j < TILE_DIM; j += BLOCK_ROWS)
    out[(y + j) * width + x] = tile[threadIdx.x][threadIdx.y + j];
}

int main(int argc, char *argv[]) {
  constexpr auto N = 1024U;
  constexpr auto mem_size = N * N * sizeof(float);

  const dim3 dimGrid(N / TILE_DIM, N / TILE_DIM);
  const dim3 dimBlock(TILE_DIM, BLOCK_ROWS);

  const auto devId = argc > 1 ? std::stoi(argv[1]) : 0;

  hipDeviceProp_t prop;
  CHECK_CUDA_ERROR(hipGetDeviceProperties(&prop, devId));
  std::printf("\nDevice : %s\n", prop.name);
  std::printf("Matrix size: %d %d, Block size: %d %d, Tile size: %d %d\n", N, N,
              TILE_DIM, BLOCK_ROWS, TILE_DIM, TILE_DIM);
  std::printf("dimGrid: %d %d %d. dimBlock %d %d %d\n", dimGrid.x, dimGrid.y,
              dimGrid.z, dimBlock.x, dimBlock.y, dimBlock.z);
  CHECK_CUDA_ERROR(hipSetDevice(devId));

  std::vector<float> host_input_data(N * N);
  std::vector<float> host_copy_data(N * N);
  std::vector<float> host_transpose_data(N * N);
  std::vector<float> host_transpose_ref_data(N * N);

  float *dev_input_data, *dev_copy_data, *dev_transpose_data;
  CHECK_CUDA_ERROR(hipMalloc(&dev_input_data, mem_size));
  CHECK_CUDA_ERROR(hipMalloc(&dev_copy_data, mem_size));
  CHECK_CUDA_ERROR(hipMalloc(&dev_transpose_data, mem_size));

  // check parameters and calculate execution configuration
  static_assert(N % TILE_DIM == 0);
  static_assert(TILE_DIM % BLOCK_ROWS == 0);

  for (int i = 0; i < N; ++i) {
    for (int j = 0; j < N; ++j) {
      const auto tmp = i * N + j;
      host_input_data[i * N + j] = tmp;
      host_transpose_ref_data[j * N + i] = tmp;
    }
  }
  CHECK_CUDA_ERROR(hipMemcpy(dev_input_data, host_input_data.data(), mem_size,
                              hipMemcpyDefault));

  float ms;
  hipEvent_t start_event, stop_event;
  CHECK_CUDA_ERROR(hipEventCreate(&start_event));
  CHECK_CUDA_ERROR(hipEventCreate(&stop_event));

  // copy
  std::printf("%25s", "copy");
  CHECK_CUDA_ERROR(hipMemset(dev_copy_data, 0, mem_size));
  copy<<<dimGrid, dimBlock>>>(dev_copy_data, dev_input_data);
  CHECK_CUDA_ERROR(hipEventRecord(start_event, 0));
  for (int i = 0; i < NUM_REPS; ++i)
    copy<<<dimGrid, dimBlock>>>(dev_copy_data, dev_input_data);
  CHECK_CUDA_ERROR(hipEventRecord(stop_event, 0));
  CHECK_CUDA_ERROR(hipEventSynchronize(stop_event));
  CHECK_CUDA_ERROR(hipEventElapsedTime(&ms, start_event, stop_event));
  CHECK_CUDA_ERROR(hipMemcpy(host_copy_data.data(), dev_copy_data, mem_size,
                              hipMemcpyDefault));
  postprocess(host_input_data.data(), host_copy_data.data(), N * N, ms);

  // copySharedMem
  std::printf("%25s", "shared memory copy");
  CHECK_CUDA_ERROR(hipMemset(dev_copy_data, 0, mem_size));
  copySharedMem<<<dimGrid, dimBlock>>>(dev_copy_data, dev_input_data);
  CHECK_CUDA_ERROR(hipEventRecord(start_event, 0));
  for (int i = 0; i < NUM_REPS; ++i)
    copySharedMem<<<dimGrid, dimBlock>>>(dev_copy_data, dev_input_data);
  CHECK_CUDA_ERROR(hipEventRecord(stop_event, 0));
  CHECK_CUDA_ERROR(hipEventSynchronize(stop_event));
  CHECK_CUDA_ERROR(hipEventElapsedTime(&ms, start_event, stop_event));
  CHECK_CUDA_ERROR(hipMemcpy(host_copy_data.data(), dev_copy_data, mem_size,
                              hipMemcpyDefault));
  postprocess(host_input_data.data(), host_copy_data.data(), N * N, ms);

  // transposeNaive
  std::printf("%25s", "naive transpose");
  CHECK_CUDA_ERROR(hipMemset(dev_transpose_data, 0, mem_size));
  transposeNaive<<<dimGrid, dimBlock>>>(dev_transpose_data, dev_input_data);
  CHECK_CUDA_ERROR(hipEventRecord(start_event, 0));
  for (int i = 0; i < NUM_REPS; ++i)
    transposeNaive<<<dimGrid, dimBlock>>>(dev_transpose_data, dev_input_data);
  CHECK_CUDA_ERROR(hipEventRecord(stop_event, 0));
  CHECK_CUDA_ERROR(hipEventSynchronize(stop_event));
  CHECK_CUDA_ERROR(hipEventElapsedTime(&ms, start_event, stop_event));
  CHECK_CUDA_ERROR(hipMemcpy(host_transpose_data.data(), dev_transpose_data,
                              mem_size, hipMemcpyDefault));
  postprocess(host_transpose_ref_data.data(), host_transpose_data.data(), N * N,
              ms);

  // transposeCoalesced
  std::printf("%25s", "coalesced tranpose");
  CHECK_CUDA_ERROR(hipMemset(dev_transpose_data, 0, mem_size));
  transposeCoalesced<<<dimGrid, dimBlock>>>(dev_transpose_data, dev_input_data);
  CHECK_CUDA_ERROR(hipEventRecord(start_event, 0));
  for (int i = 0; i < NUM_REPS; ++i)
    transposeCoalesced<<<dimGrid, dimBlock>>>(dev_transpose_data,
                                              dev_input_data);
  CHECK_CUDA_ERROR(hipEventRecord(stop_event, 0));
  CHECK_CUDA_ERROR(hipEventSynchronize(stop_event));
  CHECK_CUDA_ERROR(hipEventElapsedTime(&ms, start_event, stop_event));
  CHECK_CUDA_ERROR(hipMemcpy(host_transpose_data.data(), dev_transpose_data,
                              mem_size, hipMemcpyDefault));
  postprocess(host_transpose_ref_data.data(), host_transpose_data.data(), N * N,
              ms);

  // transposeNoBankConflicts
  std::printf("%25s", "conflict-free transpose");
  CHECK_CUDA_ERROR(hipMemset(dev_transpose_data, 0, mem_size));
  transposeNoBankConflicts<<<dimGrid, dimBlock>>>(dev_transpose_data,
                                                  dev_input_data);
  CHECK_CUDA_ERROR(hipEventRecord(start_event, 0));
  for (int i = 0; i < NUM_REPS; ++i)
    transposeNoBankConflicts<<<dimGrid, dimBlock>>>(dev_transpose_data,
                                                    dev_input_data);
  CHECK_CUDA_ERROR(hipEventRecord(stop_event, 0));
  CHECK_CUDA_ERROR(hipEventSynchronize(stop_event));
  CHECK_CUDA_ERROR(hipEventElapsedTime(&ms, start_event, stop_event));
  CHECK_CUDA_ERROR(hipMemcpy(host_transpose_data.data(), dev_transpose_data,
                              mem_size, hipMemcpyDefault));
  postprocess(host_transpose_ref_data.data(), host_transpose_data.data(), N * N,
              ms);

  CHECK_CUDA_ERROR(hipFree(dev_input_data));
  CHECK_CUDA_ERROR(hipFree(dev_copy_data));
  CHECK_CUDA_ERROR(hipFree(dev_transpose_data));
  CHECK_CUDA_ERROR(hipEventDestroy(start_event));
  CHECK_CUDA_ERROR(hipEventDestroy(stop_event));

  return 0;
}
