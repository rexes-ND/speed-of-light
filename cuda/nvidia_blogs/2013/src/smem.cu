/*
    Link: https://developer.nvidia.com/blog/using-shared-memory-cuda-cc/
*/


#include <hip/hip_runtime.h>
#include <iostream>

#include <hip/hip_runtime_api.h>

constexpr unsigned int N{64};

__global__ void staticReverse(int *d) {
  __shared__ int s[N];
  unsigned int t{threadIdx.x};
  unsigned int trev{N - t - 1};
  s[t] = d[t];
  __syncthreads();
  d[t] = s[trev];
}

__global__ void dynamicReverse(int *d) {
  extern __shared__ int s[];
  unsigned int t{threadIdx.x};
  unsigned int trev{N - t - 1};
  s[t] = d[t];
  __syncthreads();
  d[t] = s[trev];
}

int main() {
  int a[N], arev[N], d[N];

  for (int i = 0; i < N; ++i) {
    a[i] = i;
    arev[i] = N - i - 1;
    d[i] = 0;
  }

  int *d_d{};
  hipMalloc(&d_d, N * sizeof(int));

  hipMemcpy(d_d, a, N * sizeof(int), hipMemcpyHostToDevice);
  staticReverse<<<1, N>>>(d_d);
  hipMemcpy(d, d_d, N * sizeof(int), hipMemcpyDeviceToHost);

  for (int i = 0; i < N; ++i)
    if (d[i] != arev[i])
      std::printf("Error: d[%d] != arev[%d] (%d, %d)\n", i, i, d[i], arev[i]);

  hipMemcpy(d_d, a, N * sizeof(int), hipMemcpyHostToDevice);
  dynamicReverse<<<1, N, N * sizeof(int)>>>(d_d);
  hipMemcpy(d, d_d, N * sizeof(int), hipMemcpyDeviceToHost);
  for (int i = 0; i < N; ++i)
    if (d[i] != arev[i])
      std::printf("Error: d[%d] != arev[%d] (%d, %d)\n", i, i, d[i], arev[i]);

  return 0;
}
