
#include <hip/hip_runtime.h>
#include <iostream>
#include <string>

#include <hip/hip_runtime_api.h>

#define CHECK_CUDA_ERROR(err) __check_cuda_error(err, __FILE__, __LINE__)
static void __check_cuda_error(hipError_t err, const char *filename,
                               int line) {
  if (hipSuccess != err) {
    std::cerr << "CUDA API error: " << hipGetErrorString(err) << " from file "
              << filename << ", line " << line << std::endl;
    exit(err);
  }
}

template <typename T> __global__ void offset(T *a, int s) {
  const auto i = blockDim.x * blockIdx.x + threadIdx.x + s;
  a[i] = a[i] + 1;
}

template <typename T> __global__ void stride(T *a, int s) {
  const auto i = (blockDim.x * blockIdx.x + threadIdx.x) * s;
  a[i] = a[i] + 1;
}

template <typename T> void run_test(int dev_id, unsigned int nMB) {
  constexpr auto block_size = 256U;

  T *d_a;
  const auto N = (nMB << 20) / sizeof(T);
  CHECK_CUDA_ERROR(hipMalloc(&d_a, N * 33 * sizeof(T)));

  float ms;
  hipEvent_t start_event, stop_event;
  CHECK_CUDA_ERROR(hipEventCreate(&start_event));
  CHECK_CUDA_ERROR(hipEventCreate(&stop_event));

  std::cout << "Offset, Bandwidth (GB/s):" << std::endl;

  offset<<<N / block_size, block_size>>>(d_a, 0); // warm up
  for (int i = 0; i <= 32; ++i) {
    CHECK_CUDA_ERROR(hipMemset(d_a, 0, N * sizeof(T)));
    CHECK_CUDA_ERROR(hipEventRecord(start_event));
    offset<<<N / block_size, block_size>>>(d_a, i);
    CHECK_CUDA_ERROR(hipEventRecord(stop_event));
    CHECK_CUDA_ERROR(hipEventSynchronize(stop_event));

    CHECK_CUDA_ERROR(hipEventElapsedTime(&ms, start_event, stop_event));
    const auto bandwidth = 2 * nMB / ms;
    std::cout << i << ", " << bandwidth << std::endl;
  }

  std::cout << std::endl;
  std::cout << "Stride, Bandwidth (GB/s):" << std::endl;

  stride<<<N / block_size, block_size>>>(d_a, 1);
  for (int i = 1; i <= 32; ++i) {
    CHECK_CUDA_ERROR(hipMemset(d_a, 0, N * sizeof(T)));
    CHECK_CUDA_ERROR(hipEventRecord(start_event));
    stride<<<N / block_size, block_size>>>(d_a, i);
    CHECK_CUDA_ERROR(hipEventRecord(stop_event));
    CHECK_CUDA_ERROR(hipEventSynchronize(stop_event));

    CHECK_CUDA_ERROR(hipEventElapsedTime(&ms, start_event, stop_event));
    std::cout << i << ", " << 2 * nMB / ms << std::endl;
  }

  CHECK_CUDA_ERROR(hipEventDestroy(start_event));
  CHECK_CUDA_ERROR(hipEventDestroy(stop_event));

  hipFree(d_a);
}

int main(int argc, char *argv[]) {
  constexpr auto nMB = 256U;
  int dev_id = 0;
  bool is_fp64 = false;

  for (int i = 1; i < argc; ++i) {
    std::string arg = argv[i];
    if (arg.size() >= 4 && arg.substr(0, 4) == "dev=")
      dev_id = std::stoi(arg.substr(4));
    else if (arg == "fp64")
      is_fp64 = true;
  }

  hipDeviceProp_t prop;
  CHECK_CUDA_ERROR(hipSetDevice(dev_id));
  CHECK_CUDA_ERROR(hipGetDeviceProperties(&prop, dev_id));
  std::cout << "Device: " << prop.name << std::endl;
  std::cout << "Transfer size (MB): " << nMB << std::endl;
  std::cout << (is_fp64 ? "Double" : "Single") << " Precision" << std::endl;

  if (is_fp64)
    run_test<double>(dev_id, nMB);
  else
    run_test<float>(dev_id, nMB);

  return 0;
}
